#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "myProto.h"



__global__ void ComputeXYKernel(Point* points, double* x_vals, double* y_vals, int N, double t) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < N) {
        x_vals[i] = ((points[i].x2 - points[i].x1) / 2.0) * sin(t * 3.14 / 2.0) + (points[i].x2 + points[i].x1) / 2.0;
        y_vals[i] = points[i].a * x_vals[i] + points[i].b;
    }
}

void ComputeXYValues(Point* points, double* x_vals, double* y_vals, int N, double t) {
    Point* d_points = nullptr;
    double* d_x_vals = nullptr;
    double* d_y_vals = nullptr;

    hipMalloc((void**)&d_points, N * sizeof(Point));
    hipMalloc((void**)&d_x_vals, N * sizeof(double));
    hipMalloc((void**)&d_y_vals, N * sizeof(double));

    hipMemcpy(d_points, points, N * sizeof(Point), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    ComputeXYKernel<<<numBlocks, threadsPerBlock>>>(d_points, d_x_vals, d_y_vals, N, t);

    hipMemcpy(x_vals, d_x_vals, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y_vals, d_y_vals, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_x_vals);
    hipFree(d_y_vals);
}